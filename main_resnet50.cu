#include "hip/hip_runtime.h"
/* The MathWorks Inc. 2019*/ 

/* ResNet50 demo main.cu file with OpenCV interfaces to read and display data. */

#include "resnet50_wrapper.h"
#include "main_resnet50.h"
#include "resnet50_wrapper_terminate.h"
#include "resnet50_wrapper_initialize.h"
#include "opencv2/opencv.hpp"
#include <stdio.h>
#include <stdlib.h>

#define IMG_WIDTH 224
#define IMG_HEIGHT 224
#define IMG_CH 3
#define VID_DEV_ID -1
        
using namespace cv;
using namespace std;
       
static void main_resnet50_wrapper();

/* 
* Convert BGR data to RGB data, without this conversion the predictions 
* will be bad
*/
static void argInit_224x224x3_real32_T(real32_T *input, Mat & im)
{
    for(int j=0;j<224*224;j++)
    {
        //BGR to RGB
        input[2*224*224+j]=(float)(im.data[j*3+0]);
        input[1*224*224+j]=(float)(im.data[j*3+1]);
        input[0*224*224+j]=(float)(im.data[j*3+2]);
    }
}

int cmpfunc(const void * a, const void * b, void * r)
{
	float x =  ((float*)r)[*(int*)b] - ((float*)r)[*(int*)a] ;
	return ( x > 0 ? ceil(x) : floor(x) );
}

void top( float* r, int* top5 )
{
    int t[1000];
    for(int i=0; i<1000; i++)
        t[i]=i;
    qsort_r(t, 1000, sizeof(int), cmpfunc, r);
    top5[0]=t[0];
    top5[1]=t[1];
    top5[2]=t[2];
    top5[3]=t[3];
    top5[4]=t[4];
    return;
}

/* Write the prediction scores on the output video frame */
void writeData(float *output,  char synsetWords[1000][100], Mat & frame, float fps)
{
  int top5[5];
  top(output, top5);

  copyMakeBorder(frame, frame, 0, 0, 400, 0, BORDER_CONSTANT, CV_RGB(0,0,0));
  char strbuf[50];
  sprintf (strbuf, "%.2f FPS", fps);
  putText(frame, strbuf, Point(30,30), FONT_HERSHEY_DUPLEX , 1.0, CV_RGB(220,220,220), 1);
  sprintf(strbuf, "%4.1f%% %s", output[top5[0]]*100, synsetWords[top5[0]]);
  putText(frame, strbuf, Point(30,80), FONT_HERSHEY_DUPLEX , 1.0, CV_RGB(220,220,220), 1);
  sprintf(strbuf, "%4.1f%% %s", output[top5[1]]*100, synsetWords[top5[1]]);
  putText(frame, strbuf, Point(30,130), FONT_HERSHEY_DUPLEX , 1.0, CV_RGB(220,220,220), 1);
  sprintf(strbuf, "%4.1f%% %s", output[top5[2]]*100, synsetWords[top5[2]]);
  putText(frame, strbuf, Point(30,180), FONT_HERSHEY_DUPLEX , 1.0, CV_RGB(220,220,220), 1);
  sprintf(strbuf, "%4.1f%% %s", output[top5[3]]*100, synsetWords[top5[3]]);
  putText(frame, strbuf, Point(30,230), FONT_HERSHEY_DUPLEX , 1.0, CV_RGB(220,220,220), 1);
  sprintf(strbuf, "%4.1f%% %s", output[top5[4]]*100, synsetWords[top5[4]]);
  putText(frame, strbuf, Point(30,280), FONT_HERSHEY_DUPLEX , 1.0, CV_RGB(220,220,220), 1);

  imshow("resnet Demo", frame);
}

/* Read the class lables from the .txt file*/
int prepareSynset(char synsets[1000][100])
{
  FILE* fp1 = fopen("synsetWords_resnet50.txt", "r");
  if (fp1 == 0) return -1;
  for(int i=0; i<1000; i++)
  {
    fgets(synsets[i], 100, fp1);
    strtok(synsets[i], "\n");
  }
  fclose(fp1);
  return 0;
}

static void main_resnet50_wrapper(void)
{
  real32_T out[1000];
  static real32_T b[150528];

  char synsetWords[1000][100];
  if (prepareSynset(synsetWords) == -1)
  {
    printf("ERROR: Unable to find synsetWords_resnet50.txt\n");
    exit(0);
  }   

  Mat oFrame, cFrame;
  /* Initialize function 'resnet50_wrapper' input arguments. */
  /* Initialize function input argument 'in'. */
  /* Call the entry-point 'resnet50_wrapper'. */

  /* Create a Video capture object */
  VideoCapture cap(VID_DEV_ID);
  if(!cap.isOpened())
  {
    cout << "can't open camera" << endl;
    exit(0);
  }
  namedWindow("resnet Demo",WINDOW_NORMAL );
  resizeWindow("resnet Demo", 1000,1000);
  float fps=0;	
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);		
  
  while(1)
  {    
    cap >> oFrame;
    resize(oFrame,cFrame,Size(IMG_WIDTH,IMG_HEIGHT));
    
    /* convert from BGR to RGB*/
    argInit_224x224x3_real32_T(b,cFrame);
    hipEventRecord(start);
    
    /* call the resent predict  function*/
    resnet50_wrapper(b, out);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float milliseconds = -1.0;
    hipEventElapsedTime(&milliseconds, start, stop);
    fps = fps*.9+1000.0/milliseconds*.1;

    /* Write the prediction on the output frame */
    writeData(out, synsetWords, oFrame, fps);
    if(waitKey(1)%256 == 27 ) break; // stop when ESC key is pressed
  }
  
}

int32_T main(int32_T argc, const char * const argv[])
{
  (void)argc;
  (void)argv;
  
  /* Call the application intialize function */
  resnet50_wrapper_initialize();
  
  /* Call the resnet predict function */
  main_resnet50_wrapper();

 /* Call the application terminate function */
  resnet50_wrapper_terminate();
  return 0;
}

